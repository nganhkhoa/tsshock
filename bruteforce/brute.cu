#include "hip/hip_runtime.h"
typedef unsigned char uint8_t;
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

namespace SHA512 {
//#region https://github.com/LeFroid/sha256-512
typedef struct PaddedMsg {
  size_t length;
  uint8_t msg[2048];
} PaddedMsg;

// Swaps the byte order of the 32 bit unsigned integer x
__device__ inline void endianSwap32(uint32_t *x) {
  char *y = (char *)x;
  for (size_t low = 0, high = sizeof(uint32_t) - 1; high > low; ++low, --high) {
    y[low] ^= y[high];
    y[high] ^= y[low];
    y[low] ^= y[high];
  }
}

// Swaps the byte order of the 64 bit unsigned integer x
__device__ inline void endianSwap64(uint64_t *x) {
  char *y = (char *)x;
  for (size_t low = 0, high = sizeof(uint64_t) - 1; high > low; ++low, --high) {
    y[low] ^= y[high];
    y[high] ^= y[low];
    y[low] ^= y[high];
  }
}

// Swaps the byte order of the 128 bit unsigned integer x
__device__ inline void endianSwap128(__uint128_t *x) {
  char *y = (char *)x;
  for (size_t low = 0, high = sizeof(__uint128_t) - 1; high > low;
       ++low, --high) {
    y[low] ^= y[high];
    y[high] ^= y[low];
    y[low] ^= y[high];
  }
}

#define SHA512_MESSAGE_BLOCK_SIZE 128
// const int SHA512_HASH_SIZE = 64;
#define HASH_ARRAY_LEN 8
#define HASH_RESULT_ARRAY_LEN 4
// const unsigned long long MAX_VAL = 0xFFFFFFFFFFFFFFFFLLU;

// K: first 64 bits of the fractional parts of the cube roots of the first 80
// primes
const static uint64_t K[80] = {
    0x428A2F98D728AE22, 0x7137449123EF65CD, 0xB5C0FBCFEC4D3B2F,
    0xE9B5DBA58189DBBC, 0x3956C25BF348B538, 0x59F111F1B605D019,
    0x923F82A4AF194F9B, 0xAB1C5ED5DA6D8118, 0xD807AA98A3030242,
    0x12835B0145706FBE, 0x243185BE4EE4B28C, 0x550C7DC3D5FFB4E2,
    0x72BE5D74F27B896F, 0x80DEB1FE3B1696B1, 0x9BDC06A725C71235,
    0xC19BF174CF692694, 0xE49B69C19EF14AD2, 0xEFBE4786384F25E3,
    0x0FC19DC68B8CD5B5, 0x240CA1CC77AC9C65, 0x2DE92C6F592B0275,
    0x4A7484AA6EA6E483, 0x5CB0A9DCBD41FBD4, 0x76F988DA831153B5,
    0x983E5152EE66DFAB, 0xA831C66D2DB43210, 0xB00327C898FB213F,
    0xBF597FC7BEEF0EE4, 0xC6E00BF33DA88FC2, 0xD5A79147930AA725,
    0x06CA6351E003826F, 0x142929670A0E6E70, 0x27B70A8546D22FFC,
    0x2E1B21385C26C926, 0x4D2C6DFC5AC42AED, 0x53380D139D95B3DF,
    0x650A73548BAF63DE, 0x766A0ABB3C77B2A8, 0x81C2C92E47EDAEE6,
    0x92722C851482353B, 0xA2BFE8A14CF10364, 0xA81A664BBC423001,
    0xC24B8B70D0F89791, 0xC76C51A30654BE30, 0xD192E819D6EF5218,
    0xD69906245565A910, 0xF40E35855771202A, 0x106AA07032BBD1B8,
    0x19A4C116B8D2D0C8, 0x1E376C085141AB53, 0x2748774CDF8EEB99,
    0x34B0BCB5E19B48A8, 0x391C0CB3C5C95A63, 0x4ED8AA4AE3418ACB,
    0x5B9CCA4F7763E373, 0x682E6FF3D6B2B8A3, 0x748F82EE5DEFB2FC,
    0x78A5636F43172F60, 0x84C87814A1F0AB72, 0x8CC702081A6439EC,
    0x90BEFFFA23631E28, 0xA4506CEBDE82BDE9, 0xBEF9A3F7B2C67915,
    0xC67178F2E372532B, 0xCA273ECEEA26619C, 0xD186B8C721C0C207,
    0xEADA7DD6CDE0EB1E, 0xF57D4F7FEE6ED178, 0x06F067AA72176FBA,
    0x0A637DC5A2C898A6, 0x113F9804BEF90DAE, 0x1B710B35131C471B,
    0x28DB77F523047D84, 0x32CAAB7B40C72493, 0x3C9EBE0A15C9BEBC,
    0x431D67C49C100D4C, 0x4CC5D4BECB3E42B6, 0x597F299CFC657E2A,
    0x5FCB6FAB3AD6FAEC, 0x6C44198C4A475817};

// Utility functions
// Rotate x to the right by numBits
#define ROTR(x, numBits) ((x >> numBits) | (x << (64 - numBits)))

// Compression functions
#define Ch(x, y, z) ((x & y) ^ ((~x) & z))
#define Maj(x, y, z) ((x & y) ^ (x & z) ^ (y & z))

#define BigSigma0(x) (ROTR(x, 28) ^ ROTR(x, 34) ^ ROTR(x, 39))
#define BigSigma1(x) (ROTR(x, 14) ^ ROTR(x, 18) ^ ROTR(x, 41))

#define SmallSigma0(x) (ROTR(x, 1) ^ ROTR(x, 8) ^ (x >> 7))
#define SmallSigma1(x) (ROTR(x, 19) ^ ROTR(x, 61) ^ (x >> 6))

// SHA512 message schedule
// Calculate the Nth block of W
__device__ void W(uint64_t *w, int N, uint64_t *M) {
  uint64_t *mPtr = &M[(N * 16)];

  // printf("Message block %d : ", N);
  for (int i = 0; i < 16; ++i) {
    w[i] = *mPtr;
    ++mPtr;

    // printf("%" PRIx64 , w[i]);
  }
  // printf("\n");
  for (int i = 16; i < 80; ++i) {
    w[i] =
        SmallSigma1(w[i - 2]) + w[i - 7] + SmallSigma0(w[i - 15]) + w[i - 16];
  }
}

// Step 1:
// Preprocesses a given message of l bits.
// Appends "1" to end of msg, then k 0 bits such that l + 1 + k = 896 mod 1024
// and k is the smallest nonnegative solution to said equation. To this is
// appended the 128 bit block equal to the bit length l.
// char *preprocess(char *msg)
__device__ void preprocess(PaddedMsg &padded, const uint8_t *msg, size_t len,
                           size_t original_len) {
  // resulting msg wll be multiple of 1024 bits
  // size_t len = strlen(msg);
  if (msg == NULL || len == 0) {
    padded.length = 0;
    return;
  }

  size_t l = len * 8;
  size_t k = (896 - ((original_len * 8 + 1) % 1024)) % 1024;
  // printf("k = %zu\n", k);
  // printf("l = %zu\n", l);
  // printf("l + k + 1 = %zu bits, %zu bytes\n", (l+k+1), ((l+k+1)/8));

  padded.length = ((l + k + 1) / 8) + 16;
  // printf("padded.length = %zu\n", padded.length);
  // padded.msg = (uint8_t*) malloc(sizeof(uint8_t) * padded.length);
  memset(&padded.msg[0], 0, sizeof(padded.msg));
  for (size_t i = 0; i < len; ++i)
    padded.msg[i] = msg[i];
  // append to the binary string a 1 followed by k zeros
  padded.msg[len] = 0x80;

  // last 16 bytes reserved for length
  __uint128_t bigL = original_len * 8;
  endianSwap128(&bigL);
  memcpy(&padded.msg[padded.length - sizeof(__uint128_t)], &bigL,
         sizeof(__uint128_t));

  // printf("l = %d\n", l);
  // printf("k = %d\n", k);
  // printf("padlen = %d\n", padded.length);
  // printf("bigl = %d\n", bigL);
}

// Step 2:
// Parse the padded message into N 1024-bit blocks
// Each block separated into 64-bit words (therefore 16 per block)
// Returns an array of 8 64 bit words corresponding to the hashed value
__device__ uint64_t *getHash(uint64_t *retVal, PaddedMsg *p) {
  size_t N = p->length / SHA512_MESSAGE_BLOCK_SIZE;
  // printf("Number of blocks = %zu\n", N);

  // initial hash value
  // uint64_t h[8] = {
  //     0x6A09E667F3BCC908,
  //     0xBB67AE8584CAA73B,
  //     0x3C6EF372FE94F82B,
  //     0xA54FF53A5F1D36F1,
  //     0x510E527FADE682D1,
  //     0x9B05688C2B3E6C1F,
  //     0x1F83D9ABFB41BD6B,
  //     0x5BE0CD19137E2179
  // };

  // uint64_t h[8] = {
  //     0x22312194FC2BF72C,
  //     0x9F555FA3C84C64C2,
  //     0x2393B86B6F53B151,
  //     0x963877195940EABD,
  //     0x96283EE2A88EFFE3,
  //     0xBE5E1E2553863992,
  //     0x2B0199FC2C85B8AA,
  //     0x0EB72DDC81C52CA2
  // };

  // start of round 4
  uint64_t h[8] = {
      0x3c4215ffb335eec9,
      0x3661b34283d95946,
      0xec547cf905e9143d,
      0x1de189b9f94abb5f,
      0x4131ecd40ef66b7d,
      0xc76285fcbadbeb82,
      0xffe30063f0a91944,
      0x19736f2d66f93554,
  };


#if MACHINE_BYTE_ORDER == LITTLE_ENDIAN
  // Convert byte order of message to big endian
  uint64_t *msg = ((uint64_t *)&p->msg[0]);
  for (int i = 0; i < N * 16; ++i)
    endianSwap64(msg++);
#endif

  uint64_t w[80];
  uint64_t reg[HASH_ARRAY_LEN];
  for (size_t i = 0; i < N; ++i) {
    uint64_t T1, T2;
// initialize registers
#pragma unroll
    for (int i2 = 0; i2 < HASH_ARRAY_LEN; ++i2)
      reg[i2] = h[i2];

    W(w, i, ((uint64_t *)(p->msg)));

// Apply the SHA512 compression function to update registers
#pragma unroll
    for (int j = 0; j < 80; ++j) {
      T1 =
          reg[7] + BigSigma1(reg[4]) + Ch(reg[4], reg[5], reg[6]) + K[j] + w[j];
      T2 = BigSigma0(reg[0]) + Maj(reg[0], reg[1], reg[2]);

      reg[7] = reg[6];
      reg[6] = reg[5];
      reg[5] = reg[4];
      reg[4] = reg[3] + T1;
      reg[3] = reg[2];
      reg[2] = reg[1];
      reg[1] = reg[0];
      reg[0] = T1 + T2;
    }

// Compute the ith intermediate hash values
#pragma unroll
    for (int i = 0; i < HASH_ARRAY_LEN; ++i)
      h[i] += reg[i];
  }
  // Now the array h is the hash of the original message M
  memcpy(retVal, h, sizeof(uint64_t) * HASH_RESULT_ARRAY_LEN);
#if MACHINE_BYTE_ORDER == LITTLE_ENDIAN
  // Convert byte order of message to big endian
  uint64_t *retValPtr = retVal;
#pragma unroll
  for (int i = 0; i < HASH_RESULT_ARRAY_LEN; ++i)
    endianSwap64(retValPtr++);
#endif
  return retVal;
}

/// Wrapper for hashing methods, up to caller to free the return value
__device__ void SHA512Hash(uint64_t *result, const uint8_t *input, size_t len,
                           size_t original_len) {
  if (original_len == 0) {
    original_len = len;
  }
  PaddedMsg paddedMsg;
  preprocess(paddedMsg, input, len, original_len);
  getHash(result, &paddedMsg);
}
//#endregion
}; // namespace SHA512

namespace BigNum {
// https://github.com/indy256/codelibrary/blob/master/cpp/numeric/bigint.cpp
const int base_bits = 30;
const int base = (1 << base_bits);

// template<typename T>
// struct divmod_result {
//     T q, r;
//     __device__ divmod_result(T q,T r):q(q),r(r){}
// };

template <int BITS> struct bigint_t {
  typedef bigint_t<BITS> bigint;
  int z[(BITS + base_bits - 1) / base_bits], zn;

  // sign == 1 <==> value >= 0
  // sign == -1 <==> value < 0
  int sign;

  __device__ bigint_t(long long v = 0) : zn(0) {
    memset(z, 0, sizeof(z));
    *this = v;
  }
  __device__ bigint_t(const uint8_t *buf, size_t len) { fromBuffer(buf, len); }

  __device__ bigint &operator=(long long v) {
    sign = v < 0 ? -1 : 1;
    v *= sign;
    zn = 0;
    for (; v > 0; v = v / base)
      z[zn++] = (int)(v % base);
    return *this;
  }

  __device__ bigint &operator+=(const bigint &other) {
    if (sign == other.sign) {
      for (int i = 0, carry = 0; i < other.zn || carry; ++i) {
        if (i == zn)
          z[zn++] = 0;
        z[i] += carry + (i < other.zn ? other.z[i] : 0);
        carry = z[i] >= base;
        if (carry)
          z[i] -= base;
      }
    } else if (other != 0 ) {
      *this -= -other;
    }
    return *this;
  }

  __device__ friend bigint operator+(bigint a, const bigint &b) {
    a += b;
    return a;
  }

  __device__ bigint &operator-=(const bigint &other) {
    if (sign == other.sign) {
      if ((sign == 1 && *this >= other) || (sign == -1 && *this <= other)) {
        for (int i = 0, carry = 0; i < other.zn || carry; ++i) {
          z[i] -= carry + (i < other.zn ? other.z[i] : 0);
          carry = z[i] < 0;
          if (carry)
            z[i] += base;
        }
        trim();
      } else {
        *this = other - *this;
        this->sign = -this->sign;
      }
    } else {
      *this += -other;
    }
    return *this;
  }

  __device__ friend bigint operator-(bigint a, const bigint &b) {
    a -= b;
    return a;
  }

  __device__ bigint &operator*=(int v) {
    if (v < 0)
      sign = -sign, v = -v;
    for (int i = 0, carry = 0; i < zn || carry; ++i) {
      if (i == zn)
        z[zn++] = 0;
      long long cur = (long long)z[i] * v + carry;
      carry = (int)(cur / base);
      z[i] = (int)(cur % base);
    }
    trim();
    return *this;
  }

  __device__ bigint operator*(int v) const { return bigint(*this) *= v; }

  // __device__ static divmod_result<bigint> divmod(const bigint &a1, const
  // bigint &b1) {
  //     int norm = base / (b1.z[b1.zn-1] + 1);
  //     bigint a = a1.abs() * norm;
  //     bigint b = b1.abs() * norm;
  //     bigint q, r;
  //     q.zn=a.zn;

  //     for (int i = (int)a.zn - 1; i >= 0; i--) {
  //         r *= base;
  //         r += a.z[i];
  //         int s1 = b.zn < r.zn ? r.z[b.zn] : 0;
  //         int s2 = b.zn - 1 < r.zn ? r.z[b.zn - 1] : 0;
  //         int d = (int)(((long long)s1 * base + s2) / b.z[b.zn-1]);
  //         r -= b * d;
  //         while (r < 0)
  //             r += b, --d;
  //         q.z[i] = d;
  //     }

  //     q.sign = a1.sign * b1.sign;
  //     r.sign = a1.sign;
  //     q.trim();
  //     r.trim();
  //     return divmod_result<bigint>(q, r / norm);
  // }

  // __device__ bigint operator/(const bigint &v) const { return divmod(*this,
  // v).q; }

  __device__ bool divisible(const unsigned long long x) const {
    __uint128_t a = 0;
    for (int i = zn - 1; i >= 0; i--) {
      a = ((a * base) + z[i]) % x;
    }
    return a == 0;
  }
  __device__ bigint operator%(const bigint &b1) const {
    int norm = base / (b1.z[b1.zn - 1] + 1);
    bigint a = abs() * norm;
    bigint b = b1.abs() * norm;
    bigint r;

    for (int i = (int)a.zn - 1; i >= 0; i--) {
      r *= base;
      r += a.z[i];
      int s1 = b.zn < r.zn ? r.z[b.zn] : 0;
      int s2 = b.zn - 1 < r.zn ? r.z[b.zn - 1] : 0;
      int d = (int)(((long long)s1 * base + s2) / b.z[b.zn - 1]);
      r -= b * d;
      while (r < 0)
        r += b, --d;
    }

    r.sign = sign;
    r.trim();
    return r / norm;
  }

  __device__ void mod(bigint &r, const bigint &b1,
                      /*tmp storage values:*/ bigint &a, bigint &b,
                      bigint &tmp3) const {
    int norm = base / (b1.z[b1.zn - 1] + 1);
    a = *this;
    a.sign = 1;
    a *= norm;
    b = b1;
    b.sign = 1;
    b *= norm;
    r = 0;

    for (int i = (int)a.zn - 1; i >= 0; i--) {
      r *= base;
      r += a.z[i];
      int s1 = b.zn < r.zn ? r.z[b.zn] : 0;
      int s2 = b.zn - 1 < r.zn ? r.z[b.zn - 1] : 0;
      int d = (int)(((long long)s1 * base + s2) / b.z[b.zn - 1]);
      tmp3 = b;
      tmp3 *= d;
      r -= tmp3;
      while (r < 0)
        r += b, --d;
    }

    r.sign = sign;
    r.trim();
    r /= norm;
  }

  __device__ bigint &operator/=(int v) {
    if (v < 0)
      sign = -sign, v = -v;
    for (int i = (int)zn - 1, rem = 0; i >= 0; --i) {
      long long cur = z[i] + rem * (long long)base;
      z[i] = (int)(cur / v);
      rem = (int)(cur % v);
    }
    trim();
    return *this;
  }

  __device__ bigint operator/(int v) const { return bigint(*this) /= v; }

  __device__ int operator%(int v) const {
    if (v < 0)
      v = -v;
    int m = 0;
    for (int i = (int)zn - 1; i >= 0; --i)
      m = (int)((z[i] + m * (long long)base) % v);
    return m * sign;
  }

  __device__ bigint &operator/=(const bigint &v) {
    *this = *this / v;
    return *this;
  }

  __device__ bigint &operator%=(const bigint &v) {
    *this = *this % v;
    return *this;
  }

  __device__ bool operator<(const bigint &v) const {
    if (sign != v.sign)
      return sign < v.sign;
    if (zn != v.zn)
      return zn * sign < v.zn * v.sign;
    for (int i = (int)zn - 1; i >= 0; i--)
      if (z[i] != v.z[i])
        return z[i] * sign < v.z[i] * sign;
    return false;
  }

  __device__ bool operator>(const bigint &v) const { return v < *this; }

  __device__ bool operator<=(const bigint &v) const { return !(v < *this); }

  __device__ bool operator>=(const bigint &v) const { return !(*this < v); }

  __device__ bool operator==(const bigint &v) const {
    return sign == v.sign && z == v.z;
  }

  __device__ bool operator!=(const bigint &v) const { return !(*this == v); }

  __device__ void trim() {
    while (zn > 0 && z[zn - 1] == 0)
      zn--;
    if (zn == 0)
      sign = 1;
  }

  __device__ bool isZero() const { return zn == 0; }

  __device__ friend bigint operator-(bigint v) {
    if (v.zn != 0)
      v.sign = -v.sign;
    return v;
  }

  __device__ bigint abs() const { return sign == 1 ? *this : -*this; }

  __device__ long long longValue() const {
    long long res = 0;
    for (int i = (int)zn - 1; i >= 0; i--)
      res = res * base + z[i];
    return res * sign;
  }

  __device__ unsigned long long ullValue() const {
    unsigned long long res = 0;
    for (int i = (int)zn - 1; i >= 0; i--)
      res = res * base + z[i];
    return res;
  }

  __device__ bigint &operator*=(const bigint &v) {
    bigint tmp = *this;
    tmp.mul_simple(*this, v);
    return *this;
  }

  __device__ bigint operator*(const bigint &v) const {
    bigint res;
    mul_simple(res, v);
    return res;
  }

  __device__ void mul_simple(bigint &res, const bigint &v) const {
    res.sign = sign * v.sign;
    res.zn = (zn + v.zn);
    memset(res.z, 0, sizeof(z));
    for (int i = 0; i < zn; ++i)
      if (z[i])
        for (int j = 0, carry = 0; j < v.zn || carry; ++j) {
          long long cur =
              res.z[i + j] + (long long)z[i] * (j < v.zn ? v.z[j] : 0) + carry;
          carry = (int)(cur / base);
          res.z[i + j] = (int)(cur % base);
        }
    res.trim();
  }

  __device__ void plus_one() {
    // quick add 1 to last bit if even
    if (z[0] & 1 == 0) {
      z[0] &= 1;
      return;
    }

    for (int i = 0; i < zn; ++i) {
      for (int b = 0; b < base_bits; b++) {
        if (z[i] & (1 << b)) {
        } else {
          z[i] = ((z[i] >> b) | 1) << b;
          return;
        }
      }

      // this part is all 1, full conversion to 0
      z[i] = 0;
    }
  }

  // __device__ inline bigint powmod(uint64_t n, const bigint &MOD) const {
  //     bigint res=1, mul=*this;
  //     while(n>0) {
  //         if(n&1) res=res*mul%MOD;
  //         mul=mul*mul%MOD;
  //         n/=2;
  //     }
  //     return res;
  // }

  __device__ void toBuffer(uint8_t *buf, size_t &len) const {
    uint8_t *ptr = buf;
    int pb = 0;
    *ptr = 0;
    for (int i = 0; i < zn; i++) {
      for (int j = 0; j < base_bits; j++) {
        *ptr |= ((z[i] >> j) & 1) << pb;
        if (++pb == 8) {
          pb = 0;
          *++ptr = 0;
        }
      }
    }
    while (ptr > buf && *ptr == 0)
      ptr--;
    ptr++;
    for (uint8_t *a = buf, *b = ptr - 1; a < b; a++, b--) {
      uint8_t x = *a;
      *a = *b;
      *b = x;
    }
    len = (size_t)(ptr - buf);
  }

  __device__ bigint &fromBuffer(const uint8_t *buf, size_t len) {
    memset(z, 0, sizeof(z));
    zn = (len * 8 + base_bits - 1) / base_bits;
    int zi = 0, zb = 0;
    for (int i = len - 1; i >= 0; i--) {
      for (int j = 0; j < 8; j++) {
        z[zi] |= ((buf[i] >> j) & 1) << zb;
        if (++zb == base_bits) {
          zb = 0;
          zi++;
        }
      }
    }
    return *this;
  }
};

}; // namespace BigNum

extern "C" __global__ void brute(uint64_t *output, uint8_t *houtput,
                                 size_t p_len, const uint8_t *p_buf,
                                 size_t q_len, const uint8_t *q_buf,
                                 size_t p_inv_len, const uint8_t *p_inv_buf,
                                 uint64_t base, uint64_t kernel_batch_size) {
  uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

#define bits 6000
  BigNum::bigint_t<bits> p(p_buf, p_len);
  BigNum::bigint_t<bits> q(q_buf, q_len);
  BigNum::bigint_t<bits> p_inv(p_inv_buf, p_inv_len);

  uint64_t r_base = base + kernel_batch_size * idx;

  uint8_t hbuf[350] = {29,  134, 157, 123, 41,  115, 255, 43,  109, 227, 254,
                       239, 223, 212, 136, 52,  96,  123, 32,  11,  147, 219,
                       176, 175, 170, 1,   130, 140, 32,  12,  197, 151, 237,
                       127, 248, 21,  169, 136, 246, 149, 106, 1,   0,   3};

  uint64_t hash[HASH_RESULT_ARRAY_LEN];

  BigNum::bigint_t<bits> x_mod_q = 1; // = x_in_q = pow(2, r, q)

  BigNum::bigint_t<bits> tmp;
  BigNum::bigint_t<bits> tmp1;
  BigNum::bigint_t<bits> tmp2;
  BigNum::bigint_t<bits> tmp3;

  // calculate pow(2, r, q)
  {
    BigNum::bigint_t<bits> mul = 2;
    uint64_t n = r_base;
    while (n > 0) {
      if (n & 1) {
        // x_mod_q *= mul;
        tmp = x_mod_q;
        mul.mul_simple(x_mod_q, tmp);

        // x_mod_q %= N;
        tmp = x_mod_q;
        tmp.mod(x_mod_q, q, tmp1, tmp2, tmp3);
      }
      tmp = mul;
      tmp.mul_simple(mul, tmp);

      tmp = mul;
      tmp.mod(mul, q, tmp1, tmp2, tmp3);
      n /= 2;
    }
  }

  BigNum::bigint_t<bits> hashnum;
  uint64_t pnum = p.ullValue();
  uint64_t condition = (pnum - 1) / 2;

  const BigNum::bigint_t<bits> one = 1;
  const BigNum::bigint_t<bits> two = 2;
  size_t x_len = 0;

  BigNum::bigint_t<bits> x;

  for (int i = 0; i < kernel_batch_size; i++) {
    if (output[0] != 0) {
      return;
    }

    // x = ((x_in_q - 1) * inv_p_mod_q % q) * p + 1

    x = x_mod_q - one;
    tmp = x;
    p_inv.mul_simple(x, tmp);

    tmp = x;
    tmp.mod(x, q, tmp1, tmp2, tmp3);

    tmp = x;
    p.mul_simple(x, tmp);

    x.plus_one();

    uint64_t r = r_base + i;
    size_t new_len = 44;

    x.toBuffer(hbuf + new_len + 2, x_len);
    hbuf[new_len++] = x_len % 256;
    hbuf[new_len++] = x_len / 256;
    new_len += x_len;

    SHA512::SHA512Hash(hash, hbuf, new_len, new_len + 128 * 4);

    hashnum.fromBuffer((uint8_t *)hash, 32);
    if (hashnum.divisible(condition)) {
      printf("should return here %llu\n", r);
      output[0] = r;
      memcpy(houtput, hash, 32);
      return;
    }

    // x_in_q <<= 1
    // if x_in_q > q:
    //     x_in_q -= q

    tmp = x_mod_q;
    two.mul_simple(x_mod_q, tmp);
    if (x_mod_q > q) {
      tmp = x_mod_q;
      tmp.mod(x_mod_q, q, tmp1, tmp2, tmp3);
    }
  }
}
